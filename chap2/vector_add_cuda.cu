#include<hip/hip_runtime.h>
#include<stdio.h>
#include<malloc.h>


__global__ void vecAddKernel(float* A, float* B, float* C, int len){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len){
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int len){
    int size = sizeof(float) * len;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C, size);

    vecAddKernel<<<ceil(len/256.0), 256>>>(d_A, d_B, d_C, len);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
    int n = 270;
    float *a = (float*) malloc(sizeof(float) * n);
    float *b = (float*) malloc(sizeof(float) * n);
    float *c = (float*) malloc(sizeof(float) * n);
    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i*2;
        c[i] = 0;
    }
    vecAdd(a, b, c, n);
    for (int i = 0; i < n; i++)
    {
        printf("%.2f, ", c[i]);
    }
    printf("\n");
}
